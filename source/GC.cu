#include <rex/Utility/GC.hxx>

using namespace std;

REX_NS_BEGIN

vector<GC::MemoryPair> GC::_hostMem;
vector<void*> GC::_deviceMem;
GC GC::_instance;

// create the garbage collector instance
GC::GC()
{
}

// destroy the garbage collector instance
GC::~GC()
{
    // clear all of the device memory
    for ( auto& dm : _deviceMem )
    {
        hipFree( dm );
    }
    _deviceMem.clear();

    // clear all of the host memory
    for ( auto& hm : _hostMem )
    {
        CleanupCallback cleanup = hm.second;
        void*           data = hm.first;

        cleanup( data );
    }
    _hostMem.clear();
}

// register device memory
void GC::RegisterDeviceMemory( void* mem )
{
    _deviceMem.push_back( mem );
}

// unregister device memory
void GC::UnregisterDeviceMemory( void* mem )
{
    for ( uint32 i = 0; i < _deviceMem.size(); ++i )
    {
        if ( _deviceMem[ i ] == mem )
        {
            _deviceMem.erase( _deviceMem.begin() + i );
            break;
        }
    }
}

REX_NS_END