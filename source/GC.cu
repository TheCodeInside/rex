#include <rex/Utility/GC.hxx>

using namespace std;

REX_NS_BEGIN

vector<GC::MemoryPair> GC::_hostMem;
vector<void*> GC::_deviceMem;
GC GC::_instance;

// create the garbage collector instance
GC::GC()
{
}

// destroy the garbage collector instance
GC::~GC()
{
    // clear all of the device memory
    for ( auto& dm : _deviceMem )
    {
        hipFree( dm );
    }
    _deviceMem.clear();

    // clear all of the host memory
    for ( auto& hm : _hostMem )
    {
        CleanupCallback cleanup = hm.second;
        void*           data = hm.first;

        cleanup( data );
    }
    _hostMem.clear();
}

REX_NS_END