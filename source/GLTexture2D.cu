#include "hip/hip_runtime.h"
#include <rex/GL/GLTexture2D.hxx>
#include <rex/Utility/Logger.hxx>
#include <cuda_gl_interop.h>
#include <hip/hip_texture_types.h>

REX_NS_BEGIN

// create texture handle data
GLTexture2D::HandleData* GLTexture2D::CreateHandleData( GLContext& context, uint32 width, uint32 height )
{
    // ensure the desired context is the current one
    if ( !context.IsCurrent() )
    {
        context.MakeCurrent();
    }



    // create the handle
    HandleData* handle = new HandleData();



    // create the OpenGL texture
    glGenTextures( 1, &( handle->GLHandle ) );

    // initialize the texture to be the given size
    glBindTexture( GL_TEXTURE_2D, handle->GLHandle );
    glTexImage2D( GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr );
    //  glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_POINT );
    //  glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_POINT );
    //  glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP );
    //  glTexParameteri( GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP );
    glBindTexture( GL_TEXTURE_2D, 0 );



    // create the CUDA graphics resource
    hipError_t err = hipGraphicsGLRegisterImage( &( handle->CudaHandle ), handle->GLHandle, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone );
    if ( err != hipSuccess )
    {
        REX_DEBUG_LOG( "Failed to register GL texture. Reason: ", hipGetErrorString( err ) );
        delete handle;
        return nullptr;
    }

    // map the CUDA graphics resource to a CUDA array
    err = hipGraphicsMapResources( 1, &( handle->CudaHandle ), nullptr );
    if ( err != hipSuccess )
    {
        REX_DEBUG_LOG( "Failed to map graphics resource to array. Reason: ", hipGetErrorString( err ) );
        delete handle;
        return nullptr;
    }

    // get the mapped array
    err = hipGraphicsSubResourceGetMappedArray( &( handle->CudaArray ), handle->CudaHandle, 0, 0 );
    if ( err != hipSuccess )
    {
        REX_DEBUG_LOG( "Failed to get mapped array. Reason: ", hipGetErrorString( err ) );
        delete handle;
        return nullptr;
    }


    // create the CUDA texture reference
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
    texture<uchar4, 2, hipReadModeElementType> tex;
    tex.addressMode[ 0 ] = hipAddressModeClamp;
    tex.addressMode[ 1 ] = hipAddressModeClamp;
    tex.filterMode = hipFilterModePoint;


    // bind the CUDA array to a texture object
    err = hipBindTextureToArray( tex, handle->CudaArray, channelDesc );
    if ( err != hipSuccess )
    {
        REX_DEBUG_LOG( "Failed to bind texture to array. Reason: ", hipGetErrorString( err ) );
        delete handle;
        return nullptr;
    }



    // allocate the texture memory array
    uint_t memSize = width * height * sizeof( uchar4 );
    hipMalloc( &( handle->TextureMemory ), memSize );
    hipMemset( handle->TextureMemory, 0, memSize );



    // return the handle
    return handle;
}

// create texture
GLTexture2D::GLTexture2D( GLContext& context, uint32 width, uint32 height )
    : _handle( CreateHandleData( context, width, height ) ),
    _width( width ),
    _height( height )
{
}

// destroy texture
GLTexture2D::~GLTexture2D()
{
    if ( _handle )
    {
        // delete the texture memory
        hipFree( _handle->TextureMemory );

        // un-map the resources
        hipGraphicsUnmapResources( 1, &( _handle->CudaHandle ) );

        // unregister the resource
        hipGraphicsUnregisterResource( _handle->CudaHandle );

        // delete the OpenGL texture
        glDeleteTextures( 1, &( _handle->GLHandle ) );

        // set everything to 0
        memset( _handle, 0, sizeof( HandleData ) );

        // delete the handle data
        delete _handle;
    }
}

// get width
uint32 GLTexture2D::GetWidth() const
{
    return _width;
}

// get height
uint32 GLTexture2D::GetHeight() const
{
    return _height;
}

// get CUDA memory
uchar4* GLTexture2D::GetDeviceMemory()
{
    return _handle->TextureMemory;
}

// update GL texture
void GLTexture2D::UpdateOpenGLTexture()
{
    uint_t bufferSize = _width * _height * sizeof( uchar4 );
    hipError_t err = hipMemcpyToArray( _handle->CudaArray, 0, 0, _handle->TextureMemory, bufferSize, hipMemcpyDeviceToDevice );
}

REX_NS_END