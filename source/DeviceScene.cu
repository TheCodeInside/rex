#include "hip/hip_runtime.h"
#include "DeviceScene.hxx"

REX_NS_BEGIN

// launches the scene render kernel
void LaunchRenderKernel( const dim3& blocks, const dim3& grid, DeviceSceneData* sceneData )
{
    SceneRenderKernel<<<grid, blocks>>>( sceneData );
}

// the scene render kernel, where the magic happens
__global__ void SceneRenderKernel( DeviceSceneData* sd )
{
    // get the image coordinates
    const int32      x  = ( blockIdx.x * blockDim.x ) + threadIdx.x;
    const int32      y  = ( blockIdx.y * blockDim.y ) + threadIdx.y;
    const ViewPlane& vp = sd->ViewPlane;

    if ( x >= vp.Width || y >= vp.Height )
    {
        return;
    }

    // prepare for the tracing!!
    const Octree* octree     = sd->Octree;
    const real32  invSamples = 1.0f / vp.SampleCount;
    const real32  half       = 0.5f;
    const int32   n          = static_cast<int32>( sqrtf( vp.SampleCount ) );
    const real32  invn       = 1.0f / n;
    Color         color      = Color::Black();
    real32        t          = 0.0f;
    int32         sy         = 0;
    int32         sx         = 0;
    Ray           ray        = Ray( sd->Camera.GetPosition(), vec3( 0, 0, 1 ) );
    vec2          samplePoint;
    ShadePoint    shadePoint;

    // sample
    for ( sy = 0; sy < n; ++sy )
    {
        for ( sx = 0; sx < n; ++sx )
        {
            // get the pixel point
            samplePoint.x = x - half * vp.Width  + ( sx + half ) * invn;
            samplePoint.y = y - half * vp.Height + ( sy + half ) * invn;


            // set the ray direction
            ray.Direction = sd->Camera.GetRayDirection( samplePoint );


            // hit the objects in the scene
            const Geometry* geom = octree->QueryIntersections( ray, t, shadePoint );
            if ( geom )
            {
                shadePoint.Ray = ray;
                shadePoint.T = t;

                // add to the color if the ray hit
                const Material* mat = shadePoint.Material;
                color += mat->Shade( shadePoint, sd->Lights, sd->Octree );
            }
            else
            {
                color += sd->BackgroundColor;
            }
        }
    }


    // set the pixel!
    color *= invSamples;
    uint32 index = x + y * vp.Width;
    sd->Pixels[ index ] = color.ToUChar4();
}

REX_NS_END