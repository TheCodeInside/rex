#include "hip/hip_runtime.h"
#include <rex/Graphics/Color.hxx>
#include <rex/Math/Math.hxx>
#include <math.h>

REX_NS_BEGIN

// create a color
Color::Color()
    : Color( 0.0f, 0.0f, 0.0f )
{
}

// create a color w/ value for all
Color::Color( real32 all )
    : Color( all, all, all )
{
}

// create a color w/ r, g, and b
Color::Color( real32 r, real32 g, real32 b )
    : R( r ),
      G( g ),
      B( b )
{
}

// destroy a color
Color::~Color()
{
    R = 0.0f;
    G = 0.0f;
    B = 0.0f;
}

// linearly interpolate two colors
Color Color::Lerp( const Color& c1, const Color& c2, real32 amount )
{
    return Color( Math::Lerp( c1.R, c2.R, amount ),
                  Math::Lerp( c1.G, c2.G, amount ),
                  Math::Lerp( c1.B, c2.B, amount ) );
}

// darken a color
Color Color::Darken( const Color& color, real32 amount )
{
    return Color::Lerp( color, Color::Black(), amount );
}

// lighten a color
Color Color::Lighten( const Color& color, real32 amount )
{
    return Color::Lerp( color, Color::White(), amount );
}

// raise a color to a power
Color Color::Pow( const Color& color, real32 exp )
{
    return Color( powf( color.R, exp ),
                  powf( color.G, exp ),
                  powf( color.B, exp ) );
}

#pragma region Pre-defined Colors

// get red
Color Color::Red()
{
    return Color( 1.0f, 0.0f, 0.0f );
}

// get blue
Color Color::Blue()
{
    return Color( 0.0f, 0.0f, 1.0f );
}

// get green
Color Color::Green()
{
    return Color( 0.0f, 1.0f, 0.0f );
}

// get magenta
Color Color::Magenta()
{
    return Color( 1.0f, 0.0f, 1.0f );
}

// get yellow
Color Color::Yellow()
{
    return Color( 1.0f, 1.0f, 0.0f );
}

// get cyan
Color Color::Cyan()
{
    return Color( 0.0f, 1.0f, 1.0f );
}

// get white
Color Color::White()
{
    return Color( 1.0f );
}

// get black
Color Color::Black()
{
    return Color( 0.0f );
}

#pragma endregion

#pragma region Operators

bool Color::operator==( const Color& c ) const
{
    return ( R == c.R )
        && ( G == c.G )
        && ( B == c.B );
}

bool Color::operator!=( const Color& c ) const
{
    return !( *this == c );
}

Color Color::operator+( const Color& c ) const
{
    return Color( R + c.R,
                  G + c.G,
                  B + c.B );
}

Color Color::operator-( const Color& c ) const
{
    return Color( R - c.R,
                  G - c.G,
                  B - c.B );
}

Color Color::operator/( real32 s ) const
{
    return Color( R / s,
                  G / s,
                  B / s );
}

Color& Color::operator+=( const Color& c )
{
    R += c.R;
    G += c.G;
    B += c.B;
    return *this;
}

Color& Color::operator-=( const Color& c )
{
    R -= c.R;
    G -= c.G;
    B -= c.B;
    return *this;
}

Color& Color::operator*=( real32 s )
{
    R *= s;
    G *= s;
    B *= s;
    return *this;
}

Color& Color::operator/=( real32 s )
{
    R /= s;
    G /= s;
    B /= s;
    return *this;
}

Color operator*( const Color& c1, const Color& c2 )
{
    return Color( c1.R * c2.R,
                  c1.G * c2.G,
                  c1.B * c2.B );
}

Color operator*( const Color& c, real32 s )
{
    return Color( c.R * s,
                  c.G * s,
                  c.B * s );
}

Color operator*( real32 s, const Color& c )
{
    return Color( c.R * s,
                  c.G * s,
                  c.B * s );
}

std::ostream& operator<<( std::ostream& stream, const Color& color )
{
    stream << "{color : " << color.R << ", " << color.G << ", " << color.B << "}";

    return stream;
}

#pragma endregion

REX_NS_END