#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <rex/Rex.hxx>
#include <math.h>
#include <stdio.h>

REX_NS_BEGIN

/// <summary>
/// Contains scene data destined for a device.
/// </summary>
struct DeviceSceneData
{
    const DeviceList<Light*>* Lights;
    const AmbientLight*       AmbientLight;
    const Camera*             Camera;
    const Octree*             Octree;
    const ViewPlane*          ViewPlane;
    const Color*              BackgroundColor;
    Image*                    Image;
};

/// <summary>
/// Gets the next power of two that is higher than the given number.
/// </summary>
/// <param name="number">The number.</param>
static int32 GetNextPowerOfTwo( int32 number )
{
    real64 logBase2 = log( static_cast<real64>( number ) ) / log( 2.0 );
    uint32 power    = static_cast<uint32>( Math::Ceiling( logBase2 ) );

    int32 value = 1 << power;
    return value;
}

/// <summary>
/// The scene render kernel.
/// </summary>
/// <param name="sd">The scene data.</param>
__global__ void SceneRenderKernel( DeviceSceneData* sd )
{
    // get the image coordinates
    //int32 x = threadIdx.x;
    //int32 y = threadIdx.y;
    int32  x     = ( blockIdx.x * blockDim.x ) + threadIdx.x;
    int32  y     = ( blockIdx.y * blockDim.y ) + threadIdx.y;
    Image* image = sd->Image;

    if ( x >= image->GetWidth() || y >= image->GetHeight() )
    {
        return;
    }

    // prepare for the tracing!!
    const Color&     bgColor    = *sd->BackgroundColor;
    const Camera*    camera     = sd->Camera;
    const ViewPlane* vp         = sd->ViewPlane;
    const Octree*    octree     = sd->Octree;
    const real_t     invSamples = 1.0f / vp->SampleCount;
    const real_t     half       = real_t( 0.5 );
    const int32      n          = static_cast<int32>( sqrtf( vp->SampleCount ) );
    const real_t     invn       = 1.0 / n;
    Color            color      = Color::Black();
    real_t           t          = 0;
    int32            sy         = 0;
    int32            sx         = 0;
    Ray              ray        = Ray( camera->GetPosition(), Vector3( 0, 0, 1 ) );
    Vector2          samplePoint;
    ShadePoint       shadePoint;

    // sample
    for ( sy = 0; sy < n; ++sy )
    {
        for ( sx = 0; sx < n; ++sx )
        {
            // get the pixel point
            samplePoint.X = x - half * vp->Width  + ( sx + half ) * invn;
            samplePoint.Y = y - half * vp->Height + ( sy + half ) * invn;


            // set the ray direction
            ray.Direction = camera->GetRayDirection( samplePoint );


            // hit the objects in the scene
            const Geometry* geom = octree->QueryIntersections( ray, t, shadePoint );
            if ( geom )
            {
                shadePoint.Ray = ray;
                shadePoint.T   = t;

                // add to the color if the ray hit
                const Material* mat = shadePoint.Material;
                color += mat->Shade( shadePoint, sd->Lights, sd->Octree );
            }
            else
            {
                color += bgColor;
            }
        }
    }


    // set the image pixel!
    color *= invSamples;
    image->SetDevicePixel( x, y, color );
}

// renders the scene
void Scene::Render()
{
    // make sure the camera is up to date
    _camera.CalculateOrthonormalVectors();

    // create the host scene data
    DeviceSceneData hsd;
    hsd.Lights          = _lights;
    hsd.AmbientLight    = _ambientLight;
    hsd.Camera          = GC::DeviceAlloc<Camera>( _camera );
    hsd.Octree          = _octree;
    hsd.Image           = GC::DeviceAlloc<Image>( *_image );
    hsd.ViewPlane       = GC::DeviceAlloc<ViewPlane>( _viewPlane );
    hsd.BackgroundColor = GC::DeviceAlloc<Color>( _backgroundColor );


    // copy our image's contents over to the device
    _image->CopyHostToDevice();


    // create the device scene data (and copy from the host)
    DeviceSceneData* dsd = GC::DeviceAlloc<DeviceSceneData>( hsd );
    if ( dsd == nullptr )
    {
        return;
    }


    // prepare for the kernel
    int32 imgWidth  = GetNextPowerOfTwo( _image->GetWidth() );
    int32 imgHeight = GetNextPowerOfTwo( _image->GetHeight() );
    dim3  blocks    = dim3( 16, 16 );
    dim3  grid      = dim3( imgHeight / blocks.x + ( ( imgHeight % blocks.x ) == 0 ? 0 : 1 ),
                            imgWidth  / blocks.y + ( ( imgWidth  % blocks.y ) == 0 ? 0 : 1 ) );

    // start a timer
    Timer timer;
    timer.Start();

    // run the kernel
    SceneRenderKernel<<<grid, blocks>>>( dsd );

    // check for errors
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess )
    {
        REX_DEBUG_LOG( "Render kernel failed. Reason: ", hipGetErrorString( err ) );
        return;
    }

    // wait for the kernel to finish executing
    err = hipDeviceSynchronize();
    if ( err != hipSuccess )
    {
        REX_DEBUG_LOG( "Failed to synchronize device. Reason: ", hipGetErrorString( err ) );
        return;
    }


    timer.Stop();


    // copy our image's contents back to the host
    _image->CopyDeviceToHost();



    REX_DEBUG_LOG( "Render time: ", timer.GetElapsed(), "s (~", 1 / timer.GetElapsed(), " FPS)" );
}

REX_NS_END