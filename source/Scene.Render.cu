#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <rex/Rex.hxx>
#include <math.h>
#include <stdio.h>

REX_NS_BEGIN

/// <summary>
/// Contains scene data destined for a device.
/// </summary>
struct DeviceSceneData
{
    const DeviceList<Light*>* Lights;
    const AmbientLight*       AmbientLight;
    const Camera*             Camera;
    const Octree*             Octree;
    Image*                    Image;
    const ViewPlane*          ViewPlane;
    const Color*              BackgroundColor;
};

/// <summary>
/// Gets the next power of two that is higher than the given number.
/// </summary>
/// <param name="number">The number.</param>
static int32 GetNextPowerOfTwo( int32 number )
{
    real64 logBase2 = log( static_cast<real64>( number ) ) / log( 2.0 );
    uint32 power    = static_cast<uint32>( Math::Ceiling( logBase2 ) );

    int32 value = 1 << power;
    return value;
}

/// <summary>
/// Shadow hits objects in a scene.
/// </summary>
/// <param name="sd">The scene data.</param>
/// <param name="ray">The ray to check.</param>
/// <param name="sp">The shade point whose data should be populated.</param>
__device__ void SceneHitObjects( DeviceSceneData* sd, const Ray& ray, ShadePoint& sp )
{
    // prepare to check objects
    real_t t = 0.0;


    // only get the objects that the ray hits
    const Octree*   octree = sd->Octree;
    const Geometry* geom   = octree->QueryIntersections( ray, t );


    // iterate through the hit objects
    if ( geom && geom->Hit( ray, t, sp ) )
    {
        sp.HasHit   = true;
        sp.Ray      = ray;
        sp.Material = geom->GetMaterial();
        sp.HitPoint = ray.Origin + t * ray.Direction;
        sp.T        = t;
    }
}

/// <summary>
/// The scene render kernel.
/// </summary>
/// <param name="sd">The scene data.</param>
__global__ void SceneRenderKernel( DeviceSceneData* sd )
{
    // prepare for the tracing!!
    Ray              ray;
    Vector2          pp; // pixel sample point
    Color            color      = Color::Black();
    Image*           image      = sd->Image;
    const Camera*    camera     = sd->Camera;
    const ViewPlane* vp         = sd->ViewPlane;
    const real_t     invSamples = 1.0f / sd->ViewPlane->SampleCount;
    ShadePoint       sp         = nullptr;

    // set the ray's origin
    ray.Origin = camera->GetPosition();

    // get the image coordinates
    //int32 x = threadIdx.x;
    //int32 y = threadIdx.y;
    int32 x = ( blockIdx.x * blockDim.x ) + threadIdx.x;
    int32 y = ( blockIdx.y * blockDim.y ) + threadIdx.y;

    if ( x >= image->GetWidth() || y >= image->GetHeight() )
    {
        return;
    }

    // SAMPLE
    int32  n    = static_cast<int32>( sqrtf( sd->ViewPlane->SampleCount ) );
    real_t invn = 1.0 / n;
    for ( int32 sy = 0; sy < n; ++sy )
    {
        for ( int32 sx = 0; sx < n; ++sx )
        {
            // get the pixel point
            pp.X = x - real_t( 0.5 ) * vp->Width  + ( sx + real_t( 0.5 ) ) * invn;
            pp.Y = y - real_t( 0.5 ) * vp->Height + ( sy + real_t( 0.5 ) ) * invn;

            // set the ray direction
            ray.Direction = camera->GetRayDirection( pp );

            // hit the objects in the scene
            SceneHitObjects( sd, ray, sp );

            // add to the color if the ray hit
            if ( sp.HasHit )
            {
                sp.Ray = ray;

                const Material* mat = sp.Material;
                MaterialType    matType = mat->GetType();

                color += mat->Shade( sp, sd->Lights, sd->Octree );
            }
            else
            {
                color += *( sd->BackgroundColor );
            }
        }
    }


    // set the image pixel!
    color *= invSamples;
    image->SetDevicePixel( x, y, color );
}

// renders the scene
void Scene::Render()
{
    Logger::Log( "Rendering scene..." );



    // create the host scene data
    DeviceSceneData hsd;
    hsd.Lights          = _lights;
    hsd.AmbientLight    = _ambientLight;
    hsd.Camera          = GC::DeviceAlloc<Camera>( _camera );
    hsd.Octree          = _octree;
    hsd.Image           = GC::DeviceAlloc<Image>( *( _image.get() ) );
    hsd.ViewPlane       = GC::DeviceAlloc<ViewPlane>( _viewPlane );
    hsd.BackgroundColor = GC::DeviceAlloc<Color>( _backgroundColor );


    // copy our image's contents over to the device
    _image->CopyHostToDevice();


    // create the device scene data (and copy from the host)
    DeviceSceneData* dsd = GC::DeviceAlloc<DeviceSceneData>( hsd );
    if ( dsd == nullptr )
    {
        return;
    }


    // prepare for the kernel
    int32 imgWidth  = GetNextPowerOfTwo( _image->GetWidth() );
    int32 imgHeight = GetNextPowerOfTwo( _image->GetHeight() );
    dim3  blocks    = dim3( 4, 4 );
    dim3  grid      = dim3( imgHeight / blocks.x + ( ( imgHeight % blocks.x ) == 0 ? 0 : 1 ),
                            imgWidth  / blocks.y + ( ( imgWidth  % blocks.y ) == 0 ? 0 : 1 ) );

    // start a timer
    Timer timer;
    timer.Start();

    // run the kernel
    SceneRenderKernel<<<grid, blocks>>>( dsd );

    // check for errors
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess )
    {
        REX_DEBUG_LOG( "  Render kernel failed. Reason: ", hipGetErrorString( err ) );
        return;
    }

    // wait for the kernel to finish executing
    err = hipDeviceSynchronize();
    if ( err != hipSuccess )
    {
        REX_DEBUG_LOG( "  Failed to synchronize device. Reason: ", hipGetErrorString( err ) );
        return;
    }


    timer.Stop();


    // copy our image's contents back to the host
    _image->CopyDeviceToHost();



    Logger::Log( "  Done rendering." );
    Logger::Log( "  Kernel time: ", timer.GetElapsed(), " seconds" );
}

REX_NS_END