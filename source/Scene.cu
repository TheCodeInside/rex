#include "hip/hip_runtime.h"
#include <rex/Rex.hxx>

REX_NS_BEGIN

/// <summary>
/// Defines a set of scene cleanup data.
/// </summary>
struct SceneCleanupData
{
    DeviceList<Light*>*    Lights;
    AmbientLight*          AmbientLight;
    DeviceList<Geometry*>* Geometry;
    Octree*                Octree;
};


/// <summary>
/// The scene cleanup kernel.
/// </summary>
/// <param name="data">The data to cleanup.</param>
__global__ void SceneCleanupKernel( SceneCleanupData* data )
{
    if ( data->Geometry )
    {
        for ( uint_t i = 0; i < data->Geometry->GetSize(); ++i )
        {
            Geometry* geom = data->Geometry->operator[]( i );
            delete    geom;
        }
        delete data->Geometry;
    }

    if ( data->Lights )
    {
        for ( uint_t i = 0; i < data->Lights->GetSize(); ++i )
        {
            Light* light = data->Lights->operator[]( i );
            delete light;
        }
        delete data->Lights;
    }

    if ( data->AmbientLight )
    {
        delete data->AmbientLight;
    }

    if ( data->Octree )
    {
        delete data->Octree;
    }
}


// create a new scene
Scene::Scene()
    : _lights  ( nullptr ),
      _geometry( nullptr ),
      _octree  ( nullptr ),
      _texture ( nullptr )
{
}

// destroy this scene
Scene::~Scene()
{
    REX_DEBUG_LOG( "Cleaning up scene..." );


    // delete the OpenGL texture
    if ( _texture )
    {
        delete _texture;
        _texture = nullptr;
    }



    // prepare to call the cleanup kernel
    SceneCleanupData  sdHost = { _lights, _ambientLight, _geometry, _octree };
    SceneCleanupData* sdDevice = nullptr;

    // allocate and copy the cleanup information
    if ( hipSuccess != hipMalloc( (void**)( &sdDevice ), sizeof( SceneCleanupData ) ) )
    {
        REX_DEBUG_LOG( "  Failed to allocate space for data." );
        return;
    }
    if ( hipSuccess != hipMemcpy( sdDevice, &sdHost, sizeof( SceneCleanupData ), hipMemcpyHostToDevice ) )
    {
        REX_DEBUG_LOG( "  Failed to initialize device data." );
        return;
    }

    // call the kernel
    SceneCleanupKernel<<<1, 1>>>( sdDevice );

    // check for errors
    if ( hipSuccess != hipGetLastError() )
    {
        REX_DEBUG_LOG( "  Scene cleanup failed. Reason: ", hipGetErrorString( hipGetLastError() ) );
        return;
    }

    // wait for the kernel to finish executing
    if ( hipSuccess != hipDeviceSynchronize() )
    {
        REX_DEBUG_LOG( "  Failed to synchronize device. Reason: ", hipGetErrorString( hipDeviceSynchronize() ) );
        return;
    }

    // now set everything to null :D
    _lights       = nullptr;
    _ambientLight = nullptr;
    _geometry     = nullptr;
    _octree       = nullptr;


    // try to reset the device
    if ( hipSuccess != hipDeviceReset() )
    {
        REX_DEBUG_LOG( "  Failed to reset device." );
    }
}

// saves this scene's image
void Scene::SaveImage( const char* fname ) const
{
    if ( _image )
    {
        _image->Save( fname );
    }
}

// set camera position
void Scene::SetCameraPosition( const Vector3& pos )
{
    _camera.SetPosition( pos );
}

// set camera position
void Scene::SetCameraPosition( real_t x, real_t y, real_t z )
{
    _camera.SetPosition( x, y, z );
}

REX_NS_END