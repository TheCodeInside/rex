#include "hip/hip_runtime.h"
#include <rex/Utility/Image.hxx>
#include <rex/Utility/Logger.hxx>
#include <rex/Math/Math.hxx>

// include STB image write header
#pragma warning( push )
#pragma warning( disable : 4996 )
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb_image_write.h>
#pragma warning( pop )

REX_NS_BEGIN

// create image w/ width and height
Image::Image( uint16 width, uint16 height )
    : _width( width ),
      _height( height ),
      _hPixels( nullptr ),
      _dPixels( nullptr )
{
    // ensure the given sizes are okay
    if ( width > 2048 || height > 2048 )
    {
        Logger::Log( "Cannot create an image of size ", _width, "x", _height, " (max size is 2048x2048)." );
        return;
    }


    // create host pixels
    const uint32 arraySize = _width * _height;
    _hPixels = new Color[ arraySize ];

    // fill host pixels with black
    for ( uint32 i = 0; i < arraySize; ++i )
    {
        _hPixels[ i ] = Color::Black();
    }


    // create device pixels
    hipError_t err = hipMalloc( reinterpret_cast<void**>( &_dPixels ), arraySize * sizeof( Color ) );
    if ( err == hipSuccess )
    {
        // try to copy over the color data
        err = hipMemcpy( _dPixels, _hPixels, arraySize * sizeof( Color ), hipMemcpyHostToDevice );
        if ( err != hipSuccess )
        {
            // erase the memory
            hipFree( _dPixels );
            _dPixels = nullptr;

            // print out an error
            Logger::Log( "Failed to copy host image data to device." );
        }
    }
    else
    {
        _dPixels = nullptr;
        Logger::Log( "Failed to allocate space for device image." );
    }
}

// destroy image
Image::~Image()
{
    uint16* p = const_cast<uint16*>( &_width );
    *p = 0;

    p = const_cast<uint16*>( &_height );
    *p = 0;
}

// get image width
uint16 Image::GetWidth() const
{
    return _width;
}

// get image height
uint16 Image::GetHeight() const
{
    return _height;
}

// save image
bool Image::Save( const char* fname ) const
{
    // convert our floating-point colors into 8-bit colors
    std::vector<uint8> converted;
    converted.resize( _width * _height * 3 );
    size_t convInd = 0;
    for ( auto iter = _hPixels.begin(); iter != _hPixels.end(); ++iter, convInd += 3 )
    {
        Color c = *iter;
        converted[ convInd + 0 ] = static_cast<uint8>( Math::Clamp( c.R, 0.0f, 1.0f ) * 255 );
        converted[ convInd + 1 ] = static_cast<uint8>( Math::Clamp( c.G, 0.0f, 1.0f ) * 255 );
        converted[ convInd + 2 ] = static_cast<uint8>( Math::Clamp( c.B, 0.0f, 1.0f ) * 255 );
    }

    // now write out the image as a PNG
    return 0 == stbi_write_png( fname, _width, _height, 3, &( converted[ 0 ] ), _width * 3 );
}

// copy host pixels to device
void Image::CopyHostToDevice()
{
    uint32 size = _width * _height * sizeof( Color );
    memcpy( _dPixels, _hPixels, hipMemcpyHostToDevice );
}

// copy device pixels to host
void Image::CopyDeviceToHost()
{
    uint32 size = _width * _height * sizeof( Color );
    memcpy( _hPixels, _dPixels, hipMemcpyDeviceToHost );
}

// set host pixel w/ bounds checking
void Image::SetHostPixel( uint16 x, uint16 y, const Color& color )
{
    if ( x < _width && y < _height )
    {
        SetHostPixelUnchecked( x, y, color );
    }
}

// set host pixel w/o bounds checking
void Image::SetHostPixelUnchecked( uint16 x, uint16 y, const Color& color )
{
    _hPixels[ x + y * _width ] = color;
}

// set device pixel w/ bounds checking
__device__ void Image::SetDevicePixel( uint16 x, uint16 y, const Color& color )
{
    if ( x < _width && y < _height )
    {
        SetDevicePixelUnchecked( x, y, color );
    }
}

// set device pixel w/o bounds checking
__device__ void Image::SetDevicePixelUnchecked( uint16 x, uint16 y, const Color& color )
{
    _dPixels[ x + y * _width ] = color;
}

REX_NS_END