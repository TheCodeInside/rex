#include "hip/hip_runtime.h"
#include <rex/Rex.hxx>

REX_NS_BEGIN

/// <summary>
/// Defines a set of scene build data.
/// </summary>
struct SceneBuildData
{
    DeviceList<Light*>*    Lights;
    AmbientLight*          AmbientLight;
    DeviceList<Geometry*>* Geometry;
    Octree*                Octree;
};

/// <summary>
/// The scene build kernel.
/// </summary>
__global__ void SceneBuildKernel( SceneBuildData* data )
{
    // create the lists and the ambient light
    data->Lights       = new DeviceList<Light*>();
    data->Geometry     = new DeviceList<Geometry*>();
    data->AmbientLight = new AmbientLight();



    // add a directional light
    DirectionalLight* dl = new DirectionalLight();
    dl->SetDirection( Vector3( 1, 1, 2 ) );
    dl->SetRadianceScale( real_t( 1.5 ) );
    data->Lights->Add( dl );



    // prepare a material
    const real_t ka    = 0.25f;
    const real_t kd    = 0.75f;
    const real_t ks    = 0.30f;
    const real_t kpow  = 2.00f;
    const PhongMaterial white( Color::White(), ka, kd, ks, kpow );

    // add a sphere
    Sphere* sp1 = new Sphere( white, Vector3(  0.0,  0.0,  0.0 ), 10.0 );
    Sphere* sp2 = new Sphere( white, Vector3( 10.0, 10.0, 10.0 ),  4.0 );
    data->Geometry->Add( sp1 );
    data->Geometry->Add( sp2 );

    



    // calculate the min and max of the bounds
    Vector3 min, max;
    for ( uint32 i = 0; i < data->Geometry->GetSize(); ++i )
    {
        Geometry* geom = data->Geometry->operator[]( i );
        min = Vector3::Min( min, geom->GetBounds().GetMin() );
        max = Vector3::Max( max, geom->GetBounds().GetMax() );
    }

    // create the octree
    data->Octree = new Octree( min, max );

    // add the objects to the octree
    for ( uint32 i = 0; i < data->Geometry->GetSize(); ++i )
    {
        Geometry* geom = data->Geometry->operator[]( i );
        data->Octree->Add( geom );
    }
}

// build the scene
bool Scene::Build( uint16 width, uint16 height )
{
    Logger::Log( "Building scene..." );

    // make sure the image isn't too large
    if ( width > 1024 || height > 1024 )
    {
        Logger::Log( "  Image is too large. Max dimensions are 1024x1024, given ", width, "x", height, "." );
        return false;
    }

    // create the image
    _image.reset( new Image( width, height ) );

    // set the background color
    _backgroundColor = Color( real_t( 0.392157 ),
                              real_t( 0.584314 ),
                              real_t( 0.929412 ) ); // cornflower blue ;)

    // setup the view plane
    _viewPlane.Width        = width;
    _viewPlane.Height       = height;
    _viewPlane.Gamma        = 1.0f;
    _viewPlane.InvGamma     = 1.0f / _viewPlane.Gamma;
    _viewPlane.SampleCount  = 9;


    
    // prepare for calling the kernel
    Logger::Log( "  Preparing for build kernel..." );
    SceneBuildData  sdHost   = { nullptr, nullptr, nullptr };
    SceneBuildData* sdDevice = nullptr;
    if ( hipSuccess != hipMalloc( (void**)( &sdDevice ), sizeof( SceneBuildData ) ) )
    {
        Logger::Log( "  Failed to allocate space for scene data." );
        return false;
    }
    if ( hipSuccess != hipMemcpy( sdDevice, &sdHost, sizeof( SceneBuildData ), hipMemcpyHostToDevice ) )
    {
        Logger::Log( "  Failed to initialize device scene data." );
        return false;
    }

    // call the kernel
    SceneBuildKernel<<<1, 1 >>>( sdDevice );

    // check for errors
    if ( hipSuccess != hipGetLastError() )
    {
        Logger::Log( "  Scene build failed. Reason: ", hipGetErrorString( hipGetLastError() ) );
        return false;
    }

    // wait for the kernel to finish executing
    if ( hipSuccess != hipDeviceSynchronize() )
    {
        Logger::Log( "  Failed to synchronize device. Reason: ", hipGetErrorString( hipDeviceSynchronize() ) );
        return false;
    }

    // copy our data back
    if ( hipSuccess != hipMemcpy( &sdHost, sdDevice, sizeof( SceneBuildData ), hipMemcpyDeviceToHost ) )
    {
        Logger::Log( "  Failed to copy data from device." );
        return false;
    }

    // set our references
    _lights       = sdHost.Lights;
    _ambientLight = sdHost.AmbientLight;
    _geometry     = sdHost.Geometry;
    _octree       = sdHost.Octree;





    // configure the camera
    _camera.SetPosition( 0.0, 0.0, 100.0 );
    _camera.SetTarget( 0.0, 0.0, 0.0 );
    _camera.SetUp( 0.0, 1.0, 0.0 );
    _camera.SetViewPlaneDistance( 2000.0 );
    _camera.CalculateOrthonormalVectors();

    return true;
}

REX_NS_END