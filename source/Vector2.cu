#include "hip/hip_runtime.h"
#include <rex/Math/Vector2.hxx>
#include <math.h>

REX_NS_BEGIN

// create 2D vector
Vector2::Vector2()
    : Vector2( 0.0, 0.0 )
{
}

// create 2D vector
Vector2::Vector2( real_t all )
    : Vector2( all, all )
{
}

// create 2D vector
Vector2::Vector2( real_t x, real_t y )
    : X( x ),
      Y( y )
{
}

// destroy 2D vector
Vector2::~Vector2()
{
    X = Y = 0.0;
}

// get length
real_t Vector2::Length() const
{
    return std::sqrt( LengthSq() );
}

// get length squared
real_t Vector2::LengthSq() const
{
    return ( X * X ) + ( Y * Y );
}

// normalize a vector
Vector2 Vector2::Normalize( const Vector2& vec )
{
    real_t invlen = real_t( 1.0 ) / vec.Length();
    return Vector2( vec.X * invlen,
                    vec.Y * invlen );
}

// check for equality
bool Vector2::operator==( const Vector2& v ) const
{
    return ( X == v.X )
        && ( Y == v.Y );
}

// check for inequality
bool Vector2::operator!=( const Vector2& v ) const
{
    return ( X != v.X )
        || ( Y != v.Y );
}

// add two vectors
Vector2 Vector2::operator+( const Vector2& v ) const
{
    return Vector2( X + v.X,
                    Y + v.Y );
}

// subtract two vectors
Vector2 Vector2::operator-( const Vector2& v ) const
{
    return Vector2( X - v.X,
                    Y - v.Y );
}

// negate a vector
Vector2 Vector2::operator-() const
{
    return Vector2( -X, -Y );
}

// add assign a vector
Vector2& Vector2::operator+=( const Vector2& v )
{
    X += v.X;
    Y += v.Y;
    return *this;
}

// subtract assign a vector
Vector2& Vector2::operator-=( const Vector2& v )
{
    X -= v.X;
    Y -= v.Y;
    return *this;
}

// multiply assign by a scalar
Vector2& Vector2::operator*=( real_t s )
{
    X *= s;
    Y *= s;
    return *this;
}

// divide assign by a scalar
Vector2& Vector2::operator/=( real_t s )
{
    real_t invs = real_t( 1.0 ) / s;
    X *= invs;
    Y *= invs;
    return *this;
}

// multiply 2D vector and scalar
Vector2 operator*( const Vector2& v, real_t s )
{
    return Vector2( v.X * s,
                    v.Y * s );
}

// multiply 2D vector and scalar
Vector2 operator*( real_t s, const Vector2& v )
{
    return Vector2( v.X * s,
                    v.Y * s );
}

// divide 2D vector and scalar
Vector2 operator/( const Vector2& v, real_t s )
{
    real_t invs = real_t( 1.0 ) / s;
    return Vector2( v.X * invs,
                    v.Y * invs );
}

REX_NS_END