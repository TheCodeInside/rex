#include "hip/hip_runtime.h"
#include <stdio.h>
#include <rex/Rex.hxx>
#include <thread>

using namespace rex;

/// <summary>
/// Adds colors with CUDA!
/// </summary>
/// <param name="out">The color array to output to.</param>
/// <param name="lhs">The "left hand side" colors.</param>
/// <param name="rhs">The "right hand side" colors.</param>
/// <param name="colorCount">The total number of colors.</param>
hipError_t AddColorsWithCuda( Color* out, const Color* lhs, const real32* rhs, uint32 colorCount );

/// <summary>
/// The CUDA kernel for adding two colors.
/// </summary>
/// <param name="out">The color array to output to.</param>
/// <param name="lhs">The "left hand side" colors.</param>
/// <param name="rhs">The "right hand side" colors.</param>
__global__ void KernalAddColor( Color* out, const Color* lhs, const real32* rhs )
{
    int32 index = threadIdx.x;
    
    out[ index ] = rhs[ index ] * lhs[ index ];
}

int32 main( int32 argc, char** argv )
{
    const uint32 arraySize = 10;
    const Color lhs[ arraySize ] =
    {
        Color( 0.1f ),
        Color( 0.2f ),
        Color( 0.3f ),
        Color( 0.4f ),
        Color( 0.5f ),
        Color( 0.6f ),
        Color( 0.7f ),
        Color( 0.8f ),
        Color( 0.9f ),
        Color( 1.0f )
    };
    const real32 rhs[ arraySize ] =
    {
        0.01f,
        0.02f,
        0.03f,
        0.04f,
        0.05f,
        0.06f,
        0.07f,
        0.08f,
        0.09f,
        0.10f
    };
    Color out[ arraySize ];


    // Add vectors in parallel.
    hipError_t cudaStatus = AddColorsWithCuda( out, lhs, rhs, arraySize );
    if ( cudaStatus != hipSuccess )
    {
        puts( "AddColorsWithCuda failed!" );
        return 1;
    }


    // print out the colors
    for ( uint32 i = 0; i < arraySize; ++i )
    {
        const Color&  l = lhs[ i ];
        const real32& r = rhs[ i ];
        const Color&  o = out[ i ];

        printf( "{%g, %g, %g} * %g = {%g, %g, %g}\n",
                l.R, l.G, l.B,
                r,
                o.R, o.G, o.B );
    }


    // reset the device to make graphics debugging tools happy
    cudaStatus = hipDeviceReset();
    if ( cudaStatus != hipSuccess )
    {
        puts( "hipDeviceReset failed!" );
        return 1;
    }


    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t AddColorsWithCuda( Color* out, const Color* lhs, const real32* rhs, uint32 colorCount )
{
    Color*  devLhs = 0;
    real32* devRhs = 0;
    Color*  devOut = 0;
    hipError_t status;

    // select the main GPU
    status = hipSetDevice( 0 );
    if ( status != hipSuccess )
    {
        puts( "hipSetDevice failed! Do you have a CUDA-capable GPU installed?" );
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    status = hipMalloc( (void**)&devOut, colorCount * sizeof( Color ) );
    if ( status != hipSuccess )
    {
        puts( "hipMalloc failed!" );
        goto Error;
    }

    status = hipMalloc( (void**)&devLhs, colorCount * sizeof( Color ) );
    if ( status != hipSuccess )
    {
        puts( "hipMalloc failed!" );
        goto Error;
    }

    status = hipMalloc( (void**)&devRhs, colorCount * sizeof( real32 ) );
    if ( status != hipSuccess )
    {
        puts( "hipMalloc failed!" );
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    status = hipMemcpy( devLhs, lhs, colorCount * sizeof( Color ), hipMemcpyHostToDevice );
    if ( status != hipSuccess )
    {
        puts( "hipMemcpy failed!" );
        goto Error;
    }

    status = hipMemcpy( devRhs, rhs, colorCount * sizeof( real32 ), hipMemcpyHostToDevice );
    if ( status != hipSuccess )
    {
        puts( "hipMemcpy failed!" );
        goto Error;
    }

    // run our kernel with 1 thread block and one thread per color
    KernalAddColor<<<1, colorCount>>>( devOut, devLhs, devRhs );

    // ensure our kernel ran properly
    status = hipGetLastError();
    if ( status != hipSuccess )
    {
        printf( "KernalAddColor launch failed: %s\n", hipGetErrorString( status ) );
        goto Error;
    }

    // wait for the kernel to finish then check for any errors that occurred while running
    status = hipDeviceSynchronize();
    if ( status != hipSuccess )
    {
        printf( "hipDeviceSynchronize returned error code %d after launching KernalAddColor!\n", status );
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    status = hipMemcpy( out, devOut, colorCount * sizeof( Color ), hipMemcpyDeviceToHost );
    if ( status != hipSuccess )
    {
        puts( "hipMemcpy failed!" );
        goto Error;
    }

Error:
    hipFree( devOut );
    hipFree( devLhs );
    hipFree( devRhs );

    return status;
}
